#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "../common/constants.h"
#include "./cuda_computations.cuh"

int main(int argc, char **argv)
{
	int file, result_file, read_bytes = 0, write_bytes = 0, rd;
	uint8_t *src = NULL;
	int buf_size;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float elapsed_time = 0;

	if (CURRENT_TYPE == GREY)
	{
		buf_size = HEIGHT * WIDTH;
	}
	else
	{
		buf_size = HEIGHT * 3 * WIDTH;
	}

	//reading the input image
	if ((file = open(CURRENT_IMAGE_FILE, O_RDONLY)) < 0)
	{
		fprintf(stderr, "error in opening file %s\n", CURRENT_IMAGE_FILE);
		return -1;
	}

	src = (uint8_t *)malloc(buf_size * sizeof(uint8_t));

	for (rd = 0; rd < buf_size; rd += read_bytes)
	{
		if ((read_bytes = read(file, src + rd, buf_size - rd)) < 0)
		{
			return -1;
		}
	}

	close(file);

	hipEventRecord(start, 0);
	
	compute_image(src, buf_size);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("Time : %f\n", elapsed_time / 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// saving the result
	if ((result_file = open(RESULT_FILE_NAME, O_CREAT | O_WRONLY, 0644)) < 0)
	{
		fprintf(stderr, "error in opening file  %s\n", RESULT_FILE_NAME);
		return EXIT_FAILURE;
	}

	for (rd = 0; rd < buf_size; rd += write_bytes)
	{
		if ((write_bytes = write(result_file, src + rd, buf_size - rd)) < 0)
		{
			return -1;
		}
	}
	close(result_file);

	free(src);

	return 0;
}