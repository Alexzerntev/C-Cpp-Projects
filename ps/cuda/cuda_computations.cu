#include "hip/hip_runtime.h"
#include "cuda_computations.cuh"

extern "C" void compute_image(uint8_t *src, size_t buf_size)
{
	uint8_t *d_src, *d_dest, *tmp;

	hipMalloc(&d_src, buf_size * sizeof(uint8_t));
	hipMalloc(&d_dest, buf_size * sizeof(uint8_t));
	hipMemcpy(d_src, src, buf_size, hipMemcpyHostToDevice);
	hipMemset(d_dest, 0, buf_size);

	int i;
	const int block_size = 16;

	for (i = 0; i < ITERATIONS; i++)
	{
		int total_width = CURRENT_TYPE == GREY ? WIDTH : WIDTH * 3;
		int gridX = (HEIGHT + block_size - 1) / block_size;
		int gridY = (total_width + block_size - 1) / block_size;
		dim3 block(block_size, block_size);
		dim3 grid(gridX, gridY);

		if (CURRENT_TYPE == GREY)
		{
			apply_filter_grey<<<grid, block>>>(d_src, d_dest);
		}
		else
		{
			apply_filter_rgb<<<grid, block>>>(d_src, d_dest);
		}

		tmp = d_src;
		d_src = d_dest;
		d_dest = tmp;
	}

	hipGetLastError();
	hipDeviceSynchronize();

	if (ITERATIONS % 2 == 0)
	{
		hipMemcpy(src, d_src, buf_size, hipMemcpyDeviceToHost);
	}
	else
	{
		hipMemcpy(src, d_dest, buf_size, hipMemcpyDeviceToHost);
	}

	hipFree(d_src);
	hipFree(d_dest);
}

__global__ void apply_filter_grey(uint8_t *src, uint8_t *dest)
{
	int i, j, k, l;

	float blur_filter[3][3] = {{1 / 9.0 , 1 / 9.0 , 1 / 9.0}, {1 / 9.0, 1 / 9.0, 1 / 9.0}, {1 / 9.0, 1 / 9.0, 1 / 9.0}};

	size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	if (0 < x && x < HEIGHT - 1 && 0 < y && y < WIDTH - 1)
	{
		float result = 0;
		for (i = x - 1, k = 0; i <= x + 1; i++, k++)
		{
			for (j = y - 1, l = 0; j <= y + 1; j++, l++)
			{
				result += src[WIDTH * i + j] * blur_filter[k][l];
			}
		}
		dest[WIDTH * x + y] = result;
	}
}

__global__ void apply_filter_rgb(uint8_t *src, uint8_t *dest)
{
	int i, j, k, l;

	float blur_filter[3][3] = {{1 / 9.0 , 1 / 9.0 , 1 / 9.0}, {1 / 9.0, 1 / 9.0, 1 / 9.0}, {1 / 9.0, 1 / 9.0, 1 / 9.0}};

	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y;

	if (0 < x && x < HEIGHT - 1 && 0 < y && y < 3 * WIDTH - 3)
	{
		float rresult = 0, gresult = 0, bresult = 0;
		for (i = x - 1, k = 0; i <= x + 1; i++, k++)
		{
			for (j = (y * 3) - 3, l = 0; j <= (y * 3) + 3; j += 3, l++)
			{
				rresult += src[(WIDTH * 3) * i + j] * blur_filter[k][l];
				gresult += src[(WIDTH * 3) * i + j + 1] * blur_filter[k][l];
				bresult += src[(WIDTH * 3) * i + j + 2] * blur_filter[k][l];
			}
		}
		dest[WIDTH * 3 * x + (y * 3)] = rresult;
		dest[WIDTH * 3 * x + (y * 3) + 1] = gresult;
		dest[WIDTH * 3 * x + (y * 3) + 2] = bresult;
	}
}
